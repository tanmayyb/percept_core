#include "hip/hip_runtime.h"
// the usual
#include <iostream>
#include <vector>

// necessary evils
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

// include the header
#include "cf.h"

// time keeper
#include <chrono>


#define threads 256
//  NEED EDGE CONDITION HANDLER 
//  i.e. when known num_obstacles < 256




// helper functions
__device__ void __device__subtract_vectors(double* result, double* vec1, double* vec2){
  result[0] = vec1[0] - vec2[0];
  result[1] = vec1[1] - vec2[1];
  result[2] = vec1[2] - vec2[2];
}

__device__ void __device__dot_vectors(double &result, double* vec1, double *vec2){
  double product[3];
  product[0] = vec1[0] * vec2[0];
  product[1] = vec1[1] * vec2[1];
  product[2] = vec1[2] * vec2[2];
  result = product[0] + product[1] + product[2];
}

__device__ void __device__normalize_vector(double* result_vector, double* orig_vector){
  double orig_vector_mag = sqrt(orig_vector[0]*orig_vector[0] + orig_vector[1]*orig_vector[1] + orig_vector[2]*orig_vector[2]); 
  if (orig_vector_mag == 0.f){
    result_vector[0] = 0.0;
    result_vector[1] = 0.0;
    result_vector[2] = 0.0;
  }
  else{
    result_vector[0] = orig_vector[0]/orig_vector_mag;
    result_vector[1] = orig_vector[1]/orig_vector_mag;
    result_vector[2] = orig_vector[2]/orig_vector_mag;
  }
}


// fancy kernel that does everything
__global__ void circForce_kernel(
  int num_obstacles,
  Obstacle *obstacles,
  double* goalPosition,
  double* goal_vec,
  double* agentPosition,
  double* agentVelocity
){
  int i = blockIdx.x * blockDim.x + threadIdx.x;   // i refers to obstacle being computed
  if(i >= num_obstacles) return; 

  double robot_obstacle_vec[3], rel_vel[3];

  // get robot_obstacle_vec
  robot_obstacle_vec[0] = obstacles[i].getPosX() - agentPosition[0];
  robot_obstacle_vec[1] = obstacles[i].getPosY() - agentPosition[1];
  robot_obstacle_vec[2] = obstacles[i].getPosZ() - agentPosition[2];

  // get rel_vel
  rel_vel[0] = obstacles[i].getVelX() - agentVelocity[0];
  rel_vel[1] = obstacles[i].getVelY() - agentVelocity[1];
  rel_vel[2] = obstacles[i].getVelZ() - agentVelocity[2];


  // if (robot_obstacle_vec.normalized().dot(goal_vec.normalized()) < -0.01 && robot_obstacle_vec.dot(rel_vel) < -0.01) {continue;}
  double  a, b, robot_obstacle_vec_normalized[3], goal_vec_normalized[3];
  __device__normalize_vector(robot_obstacle_vec_normalized, robot_obstacle_vec);
  __device__normalize_vector(goal_vec_normalized, goal_vec);
  __device__dot_vectors(a, robot_obstacle_vec_normalized, goal_vec);
  __device__dot_vectors(b, robot_obstacle_vec, rel_vel);
  if (a < -0.01 and b < -0.01){ // compute condition
    return;
  }




  // prints
  // printf("%f %f %f\n", goalPosition[0],goalPosition[1], goalPosition[2]);

  // printf("%f %f %f\n", 
  //   obstacles[i].getPosX(),
  //   obstacles[i].getPosY(),
  //   obstacles[i].getPosZ());



} 

void launch_circForce_kernel(
    std::vector<Obstacle> *obstacles, 
    int n_obstacles,
    double k_circ, 
    double detect_shell_rad_,
    double* goalPosition,
    double* agentPosition,
    double* agentVelocity,
    double* net_force
){
    auto chrono_start = std::chrono::high_resolution_clock::now();

    const double collision_rad_ = 0.5; 
    const int active_obstacles = 0;
    const double min_obs_dist_ = detect_shell_rad_;

    std::vector<bool> known_obstacles_(n_obstacles, false);
    std::vector<double*> field_rotation_vecs_(n_obstacles*3*sizeof(double));

    // helper variables
    int obstacle_data_size = n_obstacles * sizeof(Obstacle);
    int sizeof_vector3d = 3*sizeof(double);

    // device data
    Obstacle *d_obstacles;
    double* d_goalPosition;
    double* d_agentPosition;
    double* d_agentVelocity;
    double* d_goal_vec;

    // preliminary calculations 
    // Note: can be moved inside kernel but with time cost
    double goal_vec[3];
    goal_vec[0] = goalPosition[0] - agentPosition[0];
    goal_vec[1] = goalPosition[1] - agentPosition[1];
    goal_vec[2] = goalPosition[2] - agentPosition[2];



    // alloc memory on device
    hipMalloc((void**)&d_obstacles, obstacle_data_size);
    hipMalloc((void**)&d_goalPosition, sizeof_vector3d);
    hipMalloc((void**)&d_agentPosition, sizeof_vector3d);
    hipMalloc((void**)&d_agentVelocity, sizeof_vector3d);
    hipMalloc((void**)&d_goal_vec, sizeof_vector3d);
        
    // move memory to device
    hipMemcpy(d_obstacles, (*obstacles).data(), obstacle_data_size, hipMemcpyHostToDevice);
    hipMemcpy(d_goalPosition, goalPosition, sizeof_vector3d, hipMemcpyHostToDevice);
    hipMemcpy(d_agentPosition, agentPosition, sizeof_vector3d, hipMemcpyHostToDevice);
    hipMemcpy(d_agentVelocity, agentVelocity, sizeof_vector3d, hipMemcpyHostToDevice);
    hipMemcpy(d_goal_vec, goal_vec, sizeof_vector3d, hipMemcpyHostToDevice);


    // run kernel
    int blocks = n_obstacles/threads + 1;
    circForce_kernel<<<blocks, threads>>>(
      n_obstacles,
      d_obstacles,
      d_goalPosition,
      d_goal_vec,
      d_agentPosition,
      d_agentVelocity
    );

    // synchronize
    hipDeviceSynchronize();


    // cleanup

    // prints
    auto chrono_stop = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> duration = chrono_stop - chrono_start;
    std::cout<<"\t"<<"[ detect_shell_rad_: "<<detect_shell_rad_<<", active_obstacles: "<<active_obstacles<<", duration: "<<duration.count()<<" ],"<<std::endl;

}




// best function ever
__host__  void hello_world(){
    std::cout<<"Hello World!"<<std::endl;
}