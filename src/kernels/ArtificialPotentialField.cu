#include "hip/hip_runtime.h"
// Artificial Potential Field
// credit: https://github.com/ShuiXinYun/Path_Plan/blob/master/APF_CPP/APF.h
// Khatib (1986) : https://khatib.stanford.edu/publications/pdfs/Khatib_1986_IJRR.pdf

// the usual
#include <iostream>
#include <vector>

// necessary evils
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_vector_types.h>

// include the header
#include "percept/ArtificialPotentialField.h"
#include "percept/cuda_vector_ops.cuh"

// time keeper
#include <chrono>
#include <iomanip>

#define threads 1024

namespace artificial_potential_field{
using namespace cuda_vector_ops;


__global__ void kernel(
    double3* d_net_force,
    double3* d_masses,
    size_t num_masses,
    double3 agent_position,
    double3 agent_velocity,
    double3 goal_position,
    double agent_radius,
    double mass_radius,
    double detect_shell_rad,
    double k_force
){
    extern __shared__ double3 sdata[];

    int tid = threadIdx.x;
    int i = blockIdx.x * blockDim.x + tid;

    sdata[tid] = make_double3(0.0, 0.0, 0.0); // set as zero

    // Each thread computes a force if within bound
    if (i >= num_masses) {
        return;
    }

    double3 goal_vec;
    double dist_to_goal;
    double3 mass_position;
    double3 mass_dist_vec;
    double3 mass_velocity; 
    double3 mass_rvel_vec;
    double3 force_vec;
    double3 mass_dist_vec_normalized;
    double dist_to_mass;
    double3 mass_rvel_vec_normalized;

    // implementation of obstacle heuristic circ force
    goal_vec = goal_position - agent_position;
    dist_to_goal = norm(goal_vec);
    mass_position = d_masses[i];
    mass_dist_vec = mass_position - agent_position;
    mass_velocity = make_double3(0.0, 0.0, 0.0);
    mass_rvel_vec = agent_velocity - mass_velocity;
    force_vec = make_double3(0.0, 0.0, 0.0); // set default as zero
    mass_dist_vec_normalized = normalized(mass_dist_vec);


    dist_to_mass = norm(mass_dist_vec) - (agent_radius + mass_radius);
    dist_to_mass = fmax(dist_to_mass, 1e-5); // avoid division by zero

    // implement ARTIFICIAL POTENTIAL FIELD
    if(dist_to_mass <= detect_shell_rad){ // && norm(mass_rvel_vec) > 1e-10){ 

        // calculate force vector
        force_vec = mass_dist_vec_normalized*(-1.0)*(1/dist_to_mass - 1/detect_shell_rad);
        // Khatib (1986) : https://khatib.stanford.edu/publications/pdfs/Khatib_1986_IJRR.pdf
        // Volpe and Khosla (1990): https://www.ri.cmu.edu/pub_files/pub1/volpe_r_1990_1/volpe_r_1990_1.pdf

        force_vec = force_vec * (k_force / pow(dist_to_mass, 1));      
    }




    sdata[tid] = force_vec;

reduction:
    // Perform reduction in shared memory
    __syncthreads();
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s && tid+s<num_masses) {
            sdata[tid] = sdata[tid] + sdata[tid + s];
        }
        __syncthreads();
    }

    // Thread 0 of each block adds the block's sum to the global sum using atomics
    if (tid == 0) {
        atomicAdd(&(d_net_force->x), sdata[0].x);
        atomicAdd(&(d_net_force->y), sdata[0].y);
        atomicAdd(&(d_net_force->z), sdata[0].z);
    }

}




__host__ double3 launch_kernel(
    double3* d_masses,
    size_t num_masses,
    double3 agent_position,
    double3 agent_velocity,
    double3 goal_position,
    double agent_radius,
    double mass_radius,
    double detect_shell_rad,
    double k_force, 
    double max_allowable_force,
    bool debug
){
    // Start timing if debug is enabled
    auto start_time = std::chrono::high_resolution_clock::now();

    // Allocate device memory for net force
    double3* d_net_force;
    hipError_t err = hipMalloc(&d_net_force, sizeof(double3));
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to allocate device memory: %s\n", hipGetErrorString(err));
        return make_double3(0.0, 0.0, 0.0);  // or handle error appropriately
    }
    // set memory to zero
    err = hipMemset(d_net_force, 0, sizeof(double3));
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to set device memory: %s\n", hipGetErrorString(err));
        hipFree(d_net_force);
        return make_double3(0.0, 0.0, 0.0);
    }

    int num_blocks = (num_masses + threads - 1) / threads; // ceiling division
    size_t shared_mem_size = threads * sizeof(double3);
    kernel<<<num_blocks, threads, shared_mem_size>>>(
        d_net_force, d_masses, num_masses,
        agent_position, agent_velocity, goal_position,
        agent_radius, mass_radius, detect_shell_rad,
        k_force
    ); // CUDA kernels automatically copy value-type parameters to the device when called
    // Check for kernel launch errors
    err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to launch kernel: %s\n", hipGetErrorString(err));
        hipFree(d_net_force);
        return make_double3(0.0, 0.0, 0.0);
    }

    // Add synchronization check after kernel launch
    err = hipDeviceSynchronize();
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to synchronize: %s\n", hipGetErrorString(err));
        hipFree(d_net_force);
        return make_double3(0.0, 0.0, 0.0);
    }

    // Copy result back to host
    double3 net_force;
    err = hipMemcpy(&net_force, d_net_force, sizeof(double3), hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to copy result from device: %s\n", hipGetErrorString(err));
        hipFree(d_net_force);
        return make_double3(0.0, 0.0, 0.0);
    }
    
    // Free device memory
    hipFree(d_net_force);
    // Don't free d_masses here as it was allocated elsewhere

    // cap the force magnitude
    if(max_allowable_force > 0.0){
        double force_magnitude = norm(net_force);   
        if (force_magnitude > max_allowable_force) {
            double scale = max_allowable_force / force_magnitude;
            net_force = net_force * scale;
        }
    }

    // Print timing information if debug is enabled
    if (debug) {
        auto end_time = std::chrono::high_resolution_clock::now();
        std::chrono::duration<double> elapsed = end_time - start_time;
        std::cout << std::left << std::setw(45) << "ArtificialPotentialField"
                  << "kernel execution time: " 
                  << std::fixed << std::setprecision(9) 
                  << elapsed.count() << " seconds" << std::endl;
    }

    return net_force;
}


// best function ever
__host__  void hello_cuda_world(){
  std::cout<<"Hello CUDA World! -From Artificial Potential Field Kernel <3"<<std::endl;
}


}