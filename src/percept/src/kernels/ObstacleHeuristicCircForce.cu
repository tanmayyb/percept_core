#include "hip/hip_runtime.h"
// the usual
#include <iostream>
#include <vector>

// necessary evils
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_vector_types.h>

// include the header
#include "percept/ObstacleHeuristicCircForce.h"

// time keeper
#include <chrono>


#define threads 256
//  NEED EDGE CONDITION HANDLER 
//  i.e. when known num_obstacles < 256


namespace heuristic_kernel{

__device__ inline double3 operator+(const double3& a, const double3& b) {
    return make_double3(a.x + b.x, a.y + b.y, a.z + b.z);
}

__device__ inline double3 operator-(const double3& a, const double3& b) {
    return make_double3(a.x - b.x, a.y - b.y, a.z - b.z);
}

__device__ inline double3 operator*(const double3& a, const double scalar) {
    return make_double3(a.x * scalar, a.y * scalar, a.z * scalar);
}

__device__ inline double norm(const double3 &v) {
    return v.x * v.x + v.y * v.y + v.z * v.z;
}

__device__ inline double norm_reciprocal(const double3 &v) {
    double mag2 = v.x * v.x + v.y * v.y + v.z * v.z;
    return mag2 > 0.0 ? 1.0 / sqrt(mag2) : 0.0;
}

__device__ inline double fma(double a, double b, double c) {
    return __fma_rn(a, b, c); // computes a * b + c in one instruction
}

__device__ inline double dot(const double3 &a, const double3 &b) {
    return a.x * b.x + a.y * b.y + a.z * b.z;
}

__device__ inline double3 cross(const double3 &a, const double3 &b) {
    return make_double3(a.y * b.z - a.z * b.y,
                        a.z * b.x - a.x * b.z,
                        a.x * b.y - a.y * b.x);
}

__device__ inline double3 normalize(const double3 &v) {
    double mag = sqrt(dot(v, v));
    if (mag > 0.0) {
        return v * (1.0 / mag);
    } else {
        return make_double3(0.0, 0.0, 0.0);
    }
}


// // pmaf functions
// __device__ void calculateCurrForce(
//   double* curr_force,
//   double* rot_vec,
//   double* obstacle_pos_vec, 
//   double* agent_pos_vec, 
//   double* agent_vel_vec, 
//   double* goal_pos_vec, 
//   double* rel_vel, 
//   double k_circ,
//   double dist_obs
// ){

//   double rel_vel_normalized[3];
//   double rel_vel_norm; 

//   // double vel_norm = rel_vel.norm();
//   norm(rel_vel_norm, rel_vel);

//   if(rel_vel_norm!=0.0){
//     // calculate currentVector

//     double cfagent_to_obs[3], current_vec[3], crossproduct1[3], crossproduct2[3];
//     double cfagent_to_obs_normalized[3], current_vec_normalized[3];
//     double scalar1;

//     //   Eigen::Vector3d normalized_vel = rel_vel / vel_norm;
//     normalize_vector(rel_vel_normalized, rel_vel);

//     // Eigen::Vector3d cfagent_to_obs{obstacles[obstacle_id].getPosition() - agent_pos};  
//     subtract_vectors(cfagent_to_obs, obstacle_pos_vec, agent_pos_vec);

//     // cfagent_to_obs.normalize();
//     normalize_vector(cfagent_to_obs_normalized, cfagent_to_obs);

//     // Eigen::Vector3d current{cfagent_to_obs.cross(field_rotation_vecs.at(obstacle_id))};
//     cross_vectors(current_vec, cfagent_to_obs_normalized, rot_vec);

//     // current.normalize();
//     normalize_vector(current_vec_normalized, current_vec);

//     // curr_force = (k_circ / pow(dist_obs, 2)) * rel_vel_normalized.cross(current.cross(rel_vel_normalized));
//     scalar1 = k_circ / pow(dist_obs,2);

//     cross_vectors(crossproduct1, current_vec_normalized, rel_vel_normalized);
//     cross_vectors(crossproduct2, rel_vel_normalized, crossproduct1);
//     scale_vector(curr_force, crossproduct2, scalar1);
//   }

// }



// __device__ void calculateRotationVector(
//   double* rot_vec_result,
//   int &closest_obstacle_it, 
//   int num_obstacles, 
//   ghostplanner::cfplanner::Obstacle *obstacles, 
//   int obstacle_id,
//   double* agent_pos,
//   double* goal_pos,
//   double* goal_vec
// ){

//   double dist_vec[3], obstacle_pos_vec[3], active_obstacle_pos_vec[3], dist_obs;
//   double min_dist_obs = 100.0;

//   for(int i=0; i<num_obstacles; i++){
//     if (i != obstacle_id) {
//       // double dist_obs{(obstacles[obstacle_id].getPosition() - obstacles[i].getPosition()).norm()};
//       get_obstacle_position_vector(active_obstacle_pos_vec, obstacles[obstacle_id]);
//       get_obstacle_position_vector(obstacle_pos_vec, obstacles[i]);
//       subtract_vectors(dist_vec, active_obstacle_pos_vec, obstacle_pos_vec);
//       norm(dist_obs, dist_vec);

//       if(min_dist_obs > dist_obs){
//         min_dist_obs = dist_obs;
//         closest_obstacle_it = i;
//       }
//     }
//   }

//   // printf("closest_obstacle_it: %d\n", closest_obstacle_it);

//   double obstacle_vec[3], cfagent_to_obs[3], cfagent_to_obs_normalized[3]; 
//   double cfagent_to_obs_scaled[3], dot_product1, dot_product2, current_norm;
//   double obst_current[3], goal_current[3], current_vec[3], rot_vec[3];
//   double obst_current_normalized[3], goal_current_normalized[3], current_normalized[3], rot_vec_normalized[3];

//   // Vector from active obstacle to the obstacle which is closest to the active obstacle
//   // Eigen::Vector3d obstacle_vec = obstacles[closest_obstacle_it].getPosition() - obstacles[obstacle_id].getPosition();
//   get_obstacle_position_vector(obstacle_pos_vec, obstacles[closest_obstacle_it]);
//   get_obstacle_position_vector(active_obstacle_pos_vec, obstacles[obstacle_id]);
//   subtract_vectors(obstacle_vec, obstacle_pos_vec, active_obstacle_pos_vec);

//   // Eigen::Vector3d cfagent_to_obs{obstacles[obstacle_id].getPosition() - agent_pos};
//   subtract_vectors(cfagent_to_obs, active_obstacle_pos_vec, agent_pos);

//   // cfagent_to_obs.normalize();
//   normalize_vector(cfagent_to_obs_normalized, cfagent_to_obs);

//   // Current vector is perpendicular to obstacle surface normal and shows in opposite direction of obstacle_vec
//   // Eigen::Vector3d obst_current{ (cfagent_to_obs * obstacle_vec.dot(cfagent_to_obs)) - obstacle_vec};
//   dot_vectors(dot_product1, obstacle_vec, cfagent_to_obs_normalized);
//   scale_vector(cfagent_to_obs_scaled, cfagent_to_obs_normalized, dot_product1);
//   subtract_vectors(obst_current, cfagent_to_obs_scaled, obstacle_vec);

//   // passed by kernel so we ingore: Eigen::Vector3d goal_vec{goal_pos - agent_pos};
//   // Eigen::Vector3d goal_current{goal_vec - cfagent_to_obs * (cfagent_to_obs.dot(goal_vec))};
//   dot_vectors(dot_product2, cfagent_to_obs_normalized, goal_vec);
//   scale_vector(cfagent_to_obs_scaled, cfagent_to_obs_normalized, dot_product2); // reusing cfagent_to_obs_scaled
//   subtract_vectors(goal_current, goal_vec, cfagent_to_obs_scaled);

//   // Eigen::Vector3d current{goal_current.normalized() +
//   //                         obst_current.normalized()};
//   normalize_vector(goal_current_normalized, goal_current);
//   normalize_vector(obst_current_normalized, obst_current);
//   add_vectors(current_vec, goal_current_normalized, obst_current_normalized);

//   // printf("%f\t%f\t%f\n", current_vec[0],current_vec[1],current_vec[2]);

//   // check norm
//   norm(current_norm, current_vec);
//   if (current_norm < 1e-10) {
//     current_vec[0] = 0.0;
//     current_vec[1] = 0.0;
//     current_vec[2] = 1.0;
//   }
//   normalize_vector(current_normalized, current_vec);

//   // get rotation vector
//   // Eigen::Vector3d rot_vec{current.cross(cfagent_to_obs)};
//   cross_vectors(rot_vec, current_normalized, cfagent_to_obs_normalized);

//   // rot_vec.normalize();
//   normalize_vector(rot_vec_normalized, rot_vec);

//   // return rot_vec_normalized;
//   copy_vector(rot_vec_result, rot_vec_normalized);
// }



__global__ void ObstacleHeuristic_circForce_kernel(
    double3* d_net_force,
    double3* d_masses,
    size_t num_masses,
    double3 agent_position,
    double3 agent_velocity,
    double k_circ,
    double detect_shell_rad_
){
    extern __shared__ double3 sdata[];

    int tid = threadIdx.x;
    int i = blockIdx.x * blockDim.x + tid;

    sdata[tid] = make_double3(0.0, 0.0, 0.0);

    // Each thread computes a force if within bound

    if (i>=num_masses){
        return;
    }


    // TODO: Compute your force calculation here
    // For now, just using a placeholder calculation
    double3 force = make_double3(0.0, 0.0, 0.0);
    sdata[tid] = force;





    // Perform reduction in shared memory
    __syncthreads();
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s && tid+s<num_masses) {
            sdata[tid] = sdata[tid] + sdata[tid + s];
        }
        __syncthreads();
    }

    // Thread 0 of each block adds the block's sum to the global sum using atomics
    if (tid == 0) {
        atomicAdd(&(d_net_force->x), sdata[0].x);
        atomicAdd(&(d_net_force->y), sdata[0].y);
        atomicAdd(&(d_net_force->z), sdata[0].z);
    }

}

double3 launch_ObstacleHeuristic_circForce_kernel(
    double3* d_masses,
    size_t num_masses,
    double3 agent_position,
    double3 agent_velocity,
    double k_circ, 
    double detect_shell_rad_,
    bool debug
){
    // Copy agent data to device
    double3 d_agent_position = agent_position;  
    double3 d_agent_velocity = agent_velocity;

    // Allocate device memory for net force
    double3* d_net_force;
    hipMalloc(&d_net_force, sizeof(double3));
    hipMemset(d_net_force, 0, sizeof(double3));

    // Ceiling division
    int num_blocks = (num_masses + threads - 1) / threads;  
    size_t shared_mem_size = threads * sizeof(double3);
   
    ObstacleHeuristic_circForce_kernel<<<num_blocks, threads, shared_mem_size>>>(
        d_net_force, d_masses, num_masses,
        d_agent_position, d_agent_velocity,
        k_circ, detect_shell_rad_
    );

    // Copy result back to host
    double3 net_force;
    hipMemcpy(&net_force, d_net_force, sizeof(double3), hipMemcpyDeviceToHost);
    
    // Free device memory
    hipFree(d_net_force);
    // Don't free d_masses here as it was allocated elsewhere

    return net_force;
}







// best function ever
__host__  void hello_cuda_world(){
  std::cout<<"Hello CUDA World!"<<std::endl;
}


}